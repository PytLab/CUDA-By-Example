#include "hip/hip_runtime.h"
#include "../common/cpu_bitmap.h"
#define DIM 1024

struct hipComplex 
{
    float r_;
    float i_;

    __device__ hipComplex(float r, float i) : r_(r), i_(i) {}

    __device__ float magnitude() { return r_*r_ + i_*i_; }

    __device__ hipComplex operator*(const hipComplex & a)
    {
        return hipComplex(r_*a.r_ - i_*a.i_, r_*a.i_ + i_*a.r_); 
    }

    __device__ hipComplex operator+(const hipComplex & a) { return hipComplex(r_+a.r_, i_+a.i_); }
};

__device__ int julia(int x, int y)
{
    const float scale = 1.5;
    float jx = scale*(float)(DIM/2 - x)/(DIM/2);
    float jy = scale*(float)(DIM/2 - y)/(DIM/2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    for (int i = 0; i < 200; i++)
    {
        a = a*a + c;
        if (a.magnitude() > 1000)
        {
            return 0;
        }
    }

    return 1;
}

__global__ void kernel(unsigned char* ptr)
{
    int x = blockIdx.x;
    int y = blockIdx.y;

    int offset = y*gridDim.x + x;

    int juliaValue = julia(x, y);
    ptr[offset*4] = 255*juliaValue;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}

int main()
{
    CPUBitmap bitmap(DIM, DIM);
    unsigned char* dev_bitmap; 

    hipMalloc((void**)&dev_bitmap, bitmap.image_size());

    dim3 grid(DIM, DIM);
    kernel<<<grid, 1>>>(dev_bitmap);

    hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);

    bitmap.display_and_exit();

    hipFree(dev_bitmap);

    return 0;
}

