#include "hip/hip_runtime.h"
#include <iostream>
#include "../common/book.h"

__global__ void add(int a, int b, int* c)
{
    *c = a + b;
}

int main()
{
    int c;
    int* device_c;
    hipMalloc((void**)&device_c, sizeof(int));
    add<<<1, 1>>>(2, 8, device_c);
    hipMemcpy(&c, device_c, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "2+8=" << c << std::endl;
    hipFree(device_c);

    return 0;
}

