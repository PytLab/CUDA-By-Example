
#include <hip/hip_runtime.h>
#include <iostream>

#define N 10

__global__ void add(int* a, int* b, int* c)
{
    int idx = threadIdx.x;

    if (idx < N)
    {
        c[idx] = a[idx] + b[idx];
    }
}

int main()
{
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i*i;
    }

    hipMalloc((void**)&dev_a, N*sizeof(int));
    hipMalloc((void**)&dev_b, N*sizeof(int));
    hipMalloc((void**)&dev_c, N*sizeof(int));

    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    add<<<1, N>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    for (int i = 0; i < N; i++)
    {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}

