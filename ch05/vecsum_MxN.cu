
#include <hip/hip_runtime.h>
#include <cstdio>

#define N 200

__global__ void add(int* a, int* b, int* c)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;

    if (idx < N)
    {
        c[idx] = a[idx] + b[idx];
    }
}

int main()
{
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i*i;
    }

    hipMalloc((void**)&dev_a, N*sizeof(int));
    hipMalloc((void**)&dev_b, N*sizeof(int));
    hipMalloc((void**)&dev_c, N*sizeof(int));

    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    add<<<(N+127)/128, 128>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    for (int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    return 0;
}

